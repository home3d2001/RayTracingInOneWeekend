
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"


#include <assert.h>
#include <stdio.h>

__global__ void clearPixels(float3* pixels, const unsigned int sizeX, const unsigned int sizeY) {
	 unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;   
	 unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	 pixels[y * sizeX + x] = make_float3(103 / 255.0f, 189 / 255.0f, 170 / 255.0f);
}

__global__ void randomPixels(float3* pixels, const unsigned int sizeX, const unsigned int sizeY) {
	 unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;   
	 unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	 unsigned int idx = y * sizeX + x;

	 hiprandState randState;
	 hiprand_init(idx, 0, 0, &randState);

	 pixels[idx] = make_float3(hiprand_uniform(&randState), hiprand_uniform(&randState), hiprand_uniform(&randState));
}

#define VERIFY_CUDA(x) \
{ \
	hipError_t cudaStatus = (x); \
	if (cudaStatus != hipSuccess) { \
		fprintf(stderr, "CUDA Call Failed."); \
		assert(0); \
	} \
} \

hipError_t clearImageWithCuda(float3 color, float3* hostPixels, unsigned int sizeX, unsigned int sizeY) {
	VERIFY_CUDA(hipSetDevice(0));

	size_t byteCount = sizeX * sizeY * sizeof(float3);

	float3* devicePixels = nullptr;
	VERIFY_CUDA(hipMalloc((void**)&devicePixels, byteCount));
	VERIFY_CUDA(hipMemcpy(devicePixels, hostPixels, byteCount, hipMemcpyHostToDevice));

	dim3 block(16, 16, 1);
	dim3 grid(sizeX / block.x, sizeY / block.y, 1);
	randomPixels<<<grid, block>>>(devicePixels, sizeX, sizeY);
	VERIFY_CUDA(hipGetLastError());
	VERIFY_CUDA(hipDeviceSynchronize());
	VERIFY_CUDA(hipMemcpy(hostPixels, devicePixels, byteCount, hipMemcpyDeviceToHost));

	VERIFY_CUDA(hipFree(devicePixels));

	return hipSuccess;
}

int main()
{
	const int nx = 1280;
	const int ny = 720;

	float3* pixels = new float3[nx * ny];

	hipError_t cudaStatus = clearImageWithCuda(make_float3(1.0f, 0.0f, 1.0f), pixels, nx, ny);
	if (cudaStatus != hipSuccess) {
	    fprintf(stderr, "clearImageWithCuda failed!");
        return 1;
	}

	FILE* f = fopen( "image.ppm", "w" );
	fprintf(f, "P3\n%d %d\n%d\n", nx, ny, 255);
	for (int i = 0; i < nx * ny; ++i) {
		fprintf(f, "%d %d %d ", int(255.99 * pixels[i].x),
								int(255.99 * pixels[i].y),
								int(255.99 * pixels[i].z));
	}
	fclose(f);
	delete[] pixels;

    VERIFY_CUDA(hipDeviceReset());

    return 0;
}
