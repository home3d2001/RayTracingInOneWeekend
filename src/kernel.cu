#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "cutil_math.h"
#include ""

#include <assert.h>
#include <stdio.h>

// GLAD
#include <glad/glad.h>

// GLFW
#include <GLFW/glfw3.h>

#define M_PI 3.14159265359f

struct Ray { 
	float3 o;
	float3 d;
	__device__ Ray(float3 a, float3 b) : o(a), d(b) {} 
};

__device__ float3 color(Ray& r) {
	float3 unitDirection = normalize(r.d);
	float t = 0.5f * (unitDirection.y + 1.0f);
	float3 white = make_float3(1.0f, 1.0f, 1.0f);
	float3 blue = make_float3(0.5f, 0.7f, 1.0f);
	return (1.0f - t) * white + t * blue;
}

__global__ void render(float3* pixels, const unsigned int sizeX, const unsigned int sizeY) {
	 unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;   
	 unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	 unsigned int idx = y * sizeX + x;

	 hiprandState randState;
	 hiprand_init(idx, 0, 0, &randState);

	 float3 lowerLeftCorner = make_float3(-2.0f, -1.0f, -1.0f);
	 float3 horizontal = make_float3(4.0f, 0.0f, 0.0f);
	 float3 vertical = make_float3(0.0f, 2.0f, 0.0f);
	 float3 origin = make_float3(0.0f, 0.0f, 0.0f);

	 float u = x / (float)sizeX;
	 float v = 1 - y / (float)sizeY;

	 Ray r(origin, lowerLeftCorner + u * horizontal + v * vertical);

	 pixels[idx] = color(r);
}

__global__ void clearPixels(float3* pixels, const unsigned int sizeX, const unsigned int sizeY) {
	 unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;   
	 unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	 pixels[y * sizeX + x] = make_float3(103 / 255.0f, 189 / 255.0f, 170 / 255.0f);
}

__global__ void randomPixels(float3* pixels, const unsigned int sizeX, const unsigned int sizeY) {
	 unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;   
	 unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	 unsigned int idx = y * sizeX + x;

	 hiprandState randState;
	 hiprand_init(idx, 0, 0, &randState);

	 pixels[idx] = make_float3(hiprand_uniform(&randState), hiprand_uniform(&randState), hiprand_uniform(&randState));
}

#define VERIFY_CUDA(x) \
{ \
	hipError_t cudaStatus = (x); \
	if (cudaStatus != hipSuccess) { \
		fprintf(stderr, "CUDA Call Failed."); \
		assert(0); \
	} \
} \

hipError_t clearImageWithCuda(float3 color, float3* hostPixels, unsigned int sizeX, unsigned int sizeY) {
	VERIFY_CUDA(hipSetDevice(0));

	size_t byteCount = sizeX * sizeY * sizeof(float3);

	float3* devicePixels = nullptr;
	VERIFY_CUDA(hipMalloc((void**)&devicePixels, byteCount));
	VERIFY_CUDA(hipMemcpy(devicePixels, hostPixels, byteCount, hipMemcpyHostToDevice));

	dim3 block(16, 16, 1);
	dim3 grid(sizeX / block.x, sizeY / block.y, 1);
	render<<<grid, block>>>(devicePixels, sizeX, sizeY);
	VERIFY_CUDA(hipGetLastError());
	VERIFY_CUDA(hipDeviceSynchronize());
	VERIFY_CUDA(hipMemcpy(hostPixels, devicePixels, byteCount, hipMemcpyDeviceToHost));

	VERIFY_CUDA(hipFree(devicePixels));

	return hipSuccess;
}

class FullScreenQuad {
public:
	FullScreenQuad() {
        static const char * vs_source[] =
        {
            "#version 420 core                                                 \n"
            "                                                                  \n"
			"out vec2 uv;                                                      \n"
            "                                                                  \n"
            "void main(void)                                                   \n"
            "{                                                                 \n"
			"    uv = vec2((gl_VertexID << 1) & 2, gl_VertexID & 2);           \n"
			"    gl_Position = vec4(uv * vec2(2,-2) + vec2(-1,1), 0, 1);       \n"
            "}                                                                 \n"
        };

        static const char * fs_source[] =
        {
            "#version 420 core                                                 \n"
			"in vec2 uv;                                                       \n"
            "                                                                  \n"
            "out vec4 color;                                                   \n"
            "                                                                  \n"
            "void main(void)                                                   \n"
            "{                                                                 \n"
            "    color = vec4(uv.x, uv.y, 0.0, 1.0);                           \n"
            "}                                                                 \n"
        };

        program = glCreateProgram();
        GLuint fs = glCreateShader(GL_FRAGMENT_SHADER);
        glShaderSource(fs, 1, fs_source, NULL);
        glCompileShader(fs);
		GLint params;
		glGetShaderiv(fs, GL_COMPILE_STATUS, &params);
		if (params == GL_FALSE) {
			GLint infoLogLength;
			glGetShaderiv(fs, GL_INFO_LOG_LENGTH, &infoLogLength);

			GLchar* strInfoLog = new GLchar[infoLogLength + 1];
			glGetShaderInfoLog(fs, infoLogLength, NULL, strInfoLog);
			printf("%s", strInfoLog);
			delete strInfoLog;
		}

        GLuint vs = glCreateShader(GL_VERTEX_SHADER);
        glShaderSource(vs, 1, vs_source, NULL);
        glCompileShader(vs);
		glGetShaderiv(vs, GL_COMPILE_STATUS, &params);
		if (params == GL_FALSE) {
			GLint infoLogLength;
			glGetShaderiv(vs, GL_INFO_LOG_LENGTH, &infoLogLength);

			GLchar* strInfoLog = new GLchar[infoLogLength + 1];
			glGetShaderInfoLog(vs, infoLogLength, NULL, strInfoLog);
			printf("%s", strInfoLog);
			delete strInfoLog;
		}

        glAttachShader(program, vs);
        glAttachShader(program, fs);

        glLinkProgram(program);

        glGenVertexArrays(1, &vao);
        glBindVertexArray(vao);
	}

	~FullScreenQuad() {
		glDeleteVertexArrays(1, &vao);
		glDeleteProgram(program);
	}

	void draw() {
	    static const GLfloat green[] = { 0.0f, 0.25f, 0.0f, 1.0f };
        glClearBufferfv(GL_COLOR, 0, green);

        glUseProgram(program);
        glDrawArrays(GL_TRIANGLES, 0, 3);
	}

private:
	GLuint program;
	GLuint vao;
};

int main()
{
	const int nx = 1280;
	const int ny = 720;

    if (!glfwInit())
        return -1;

	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 2);
	glfwWindowHint(GLFW_RESIZABLE, GL_FALSE);

	GLFWwindow* window = glfwCreateWindow(nx, ny, "cudaRayTracer", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

	glfwMakeContextCurrent(window);

	if (!gladLoadGLLoader((GLADloadproc) glfwGetProcAddress)) {
        return -1;
    }

	float3* pixels = new float3[nx * ny];

	hipError_t cudaStatus = clearImageWithCuda(make_float3(1.0f, 0.0f, 1.0f), pixels, nx, ny);
	if (cudaStatus != hipSuccess) {
	    fprintf(stderr, "clearImageWithCuda failed!");
        return 1;
	}

	FILE* f = fopen( "gpu.ppm", "w" );
	fprintf(f, "P3\n%d %d\n%d\n", nx, ny, 255);
	for (int i = 0; i < nx * ny; ++i) {
		fprintf(f, "%d %d %d ", int(255.99 * pixels[i].x),
								int(255.99 * pixels[i].y),
								int(255.99 * pixels[i].z));
	}
	fclose(f);
	delete[] pixels;

	FullScreenQuad quad;

    while (!glfwWindowShouldClose(window)) {
		quad.draw();
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    VERIFY_CUDA(hipDeviceReset());

	glfwTerminate();

    return 0;
}
