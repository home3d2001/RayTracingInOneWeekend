#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "cutil_math.h"
#include ""

#include <assert.h>
#include <stdio.h>

#include <GLFW/glfw3.h>

#define M_PI 3.14159265359f

struct Ray { 
	float3 o;
	float3 d;
	__device__ Ray(float3 a, float3 b) : o(a), d(b) {} 
};

__device__ float3 color(Ray& r) {
	float3 unitDirection = normalize(r.d);
	float t = 0.5f * (unitDirection.y + 1.0f);
	float3 white = make_float3(1.0f, 1.0f, 1.0f);
	float3 blue = make_float3(0.5f, 0.7f, 1.0f);
	return (1.0f - t) * white + t * blue;
}

__global__ void render(float3* pixels, const unsigned int sizeX, const unsigned int sizeY) {
	 unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;   
	 unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	 unsigned int idx = y * sizeX + x;

	 hiprandState randState;
	 hiprand_init(idx, 0, 0, &randState);

	 float3 lowerLeftCorner = make_float3(-2.0f, -1.0f, -1.0f);
	 float3 horizontal = make_float3(4.0f, 0.0f, 0.0f);
	 float3 vertical = make_float3(0.0f, 2.0f, 0.0f);
	 float3 origin = make_float3(0.0f, 0.0f, 0.0f);

	 float u = x / (float)sizeX;
	 float v = 1 - y / (float)sizeY;

	 Ray r(origin, lowerLeftCorner + u * horizontal + v * vertical);

	 pixels[idx] = color(r);
}

__global__ void clearPixels(float3* pixels, const unsigned int sizeX, const unsigned int sizeY) {
	 unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;   
	 unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	 pixels[y * sizeX + x] = make_float3(103 / 255.0f, 189 / 255.0f, 170 / 255.0f);
}

__global__ void randomPixels(float3* pixels, const unsigned int sizeX, const unsigned int sizeY) {
	 unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;   
	 unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	 unsigned int idx = y * sizeX + x;

	 hiprandState randState;
	 hiprand_init(idx, 0, 0, &randState);

	 pixels[idx] = make_float3(hiprand_uniform(&randState), hiprand_uniform(&randState), hiprand_uniform(&randState));
}

#define VERIFY_CUDA(x) \
{ \
	hipError_t cudaStatus = (x); \
	if (cudaStatus != hipSuccess) { \
		fprintf(stderr, "CUDA Call Failed."); \
		assert(0); \
	} \
} \

hipError_t clearImageWithCuda(float3 color, float3* hostPixels, unsigned int sizeX, unsigned int sizeY) {
	VERIFY_CUDA(hipSetDevice(0));

	size_t byteCount = sizeX * sizeY * sizeof(float3);

	float3* devicePixels = nullptr;
	VERIFY_CUDA(hipMalloc((void**)&devicePixels, byteCount));
	VERIFY_CUDA(hipMemcpy(devicePixels, hostPixels, byteCount, hipMemcpyHostToDevice));

	dim3 block(16, 16, 1);
	dim3 grid(sizeX / block.x, sizeY / block.y, 1);
	render<<<grid, block>>>(devicePixels, sizeX, sizeY);
	VERIFY_CUDA(hipGetLastError());
	VERIFY_CUDA(hipDeviceSynchronize());
	VERIFY_CUDA(hipMemcpy(hostPixels, devicePixels, byteCount, hipMemcpyDeviceToHost));

	VERIFY_CUDA(hipFree(devicePixels));

	return hipSuccess;
}

int main()
{
	const int nx = 1280;
	const int ny = 720;

    if (!glfwInit())
        return -1;

	float3* pixels = new float3[nx * ny];

	hipError_t cudaStatus = clearImageWithCuda(make_float3(1.0f, 0.0f, 1.0f), pixels, nx, ny);
	if (cudaStatus != hipSuccess) {
	    fprintf(stderr, "clearImageWithCuda failed!");
        return 1;
	}

	FILE* f = fopen( "gpu.ppm", "w" );
	fprintf(f, "P3\n%d %d\n%d\n", nx, ny, 255);
	for (int i = 0; i < nx * ny; ++i) {
		fprintf(f, "%d %d %d ", int(255.99 * pixels[i].x),
								int(255.99 * pixels[i].y),
								int(255.99 * pixels[i].z));
	}
	fclose(f);
	delete[] pixels;

    VERIFY_CUDA(hipDeviceReset());

    return 0;
}
